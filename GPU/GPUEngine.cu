#include "hip/hip_runtime.h"
/*
* This file is part of the BTCCollider distribution (https://github.com/JeanLucPons/Kangaroo).
* Copyright (c) 2020 Jean Luc PONS.
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, version 3.
*
* This program is distributed in the hope that it will be useful, but
* WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
* General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef WIN64
#include <unistd.h>
#include <stdio.h>
#endif

#include "GPUEngine.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdint.h>
#include "../Timer.h"

#include "GPUMath.h"
#include "GPUCompute.h"

// ---------------------------------------------------------------------------------------

__global__ void comp_kangaroos(uint64_t *kangaroos,uint32_t maxFound,uint32_t *found,uint64_t dpMask) {

  int xPtr = (blockIdx.x*blockDim.x*GPU_GRP_SIZE) * KSIZE; // x[4] , y[4] , d[2], lastJump
  ComputeKangaroos(kangaroos + xPtr,maxFound,found,dpMask);

}

// ---------------------------------------------------------------------------------------
//#define GPU_CHECK
#ifdef GPU_CHECK
__global__ void check_gpu() {

  // Check ModInv
  uint64_t N[5] = { 0x0BE3D7593BE1147CULL,0x4952AAF512875655ULL,0x08884CCAACCB9B53ULL,0x9EAE2E2225044292ULL,0ULL };
  uint64_t I[5];
  uint64_t R[5];
  bool ok = true;

  /*
  for(uint64_t i=0;i<10000 && ok;i++) {

    Load(R,N);
    _ModInv(R);
    Load(I,R);
    _ModMult(R,N);
    SubP(R);
    if(!_IsOne(R)) {
      ok = false;
      printf("ModInv wrong %d\n",(int)i);
      printf("N = %016llx %016llx %016llx %016llx %016llx\n",N[4],N[3],N[2],N[1],N[0]);
      printf("I = %016llx %016llx %016llx %016llx %016llx\n",I[4],I[3],I[2],I[1],I[0]);
      printf("R = %016llx %016llx %016llx %016llx %016llx\n",R[4],R[3],R[2],R[1],R[0]);
    }

    N[0]++;

  }
  */
  I[4] = 0;
  R[4] = 0;
  for(uint64_t i = 0; i < 100000 && ok; i++) {

    _ModSqr(I,N);
    _ModMult(R,N,N);
    if(!_IsEqual(I,R)) {
      ok = false;
      printf("_ModSqr wrong %d\n",(int)i);
      printf("N = %016llx %016llx %016llx %016llx %016llx\n",N[4],N[3],N[2],N[1],N[0]);
      printf("I = %016llx %016llx %016llx %016llx %016llx\n",I[4],I[3],I[2],I[1],I[0]);
      printf("R = %016llx %016llx %016llx %016llx %016llx\n",R[4],R[3],R[2],R[1],R[0]);
    }

    N[0]++;

  }

}
#endif

// ---------------------------------------------------------------------------------------

using namespace std;

int _ConvertSMVer2Cores(int major,int minor) {

  // Defines for GPU Architecture types (using the SM version to determine
  // the # of cores per SM
  typedef struct {
    int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
             // and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
    { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
    { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
    { 0x30, 192 },
    { 0x32, 192 },
    { 0x35, 192 },
    { 0x37, 192 },
    { 0x50, 128 },
    { 0x52, 128 },
    { 0x53, 128 },
    { 0x60,  64 },
    { 0x61, 128 },
    { 0x62, 128 },
    { 0x70,  64 },
    { 0x72,  64 },
    { 0x75,  64 },
    { -1, -1 } };

  int index = 0;

  while(nGpuArchCoresPerSM[index].SM != -1) {
    if(nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  return 0;

}

void GPUEngine::SetWildOffset(Int* offset) {
  wildOffset.Set(offset);
}

GPUEngine::GPUEngine(int nbThreadGroup,int nbThreadPerGroup,int gpuId,uint32_t maxFound) {

  // Initialise CUDA
  this->nbThreadPerGroup = nbThreadPerGroup;
  initialised = false;
  hipError_t err;

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if(error_id != hipSuccess) {
    printf("GPUEngine: CudaGetDeviceCount %s\n",hipGetErrorString(error_id));
    return;
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if(deviceCount == 0) {
    printf("GPUEngine: There are no available device(s) that support CUDA\n");
    return;
  }

  err = hipSetDevice(gpuId);
  if(err != hipSuccess) {
    printf("GPUEngine: %s\n",hipGetErrorString(err));
    return;
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp,gpuId);

  this->nbThread = nbThreadGroup * nbThreadPerGroup;
  this->maxFound = maxFound;
  this->outputSize = (maxFound*ITEM_SIZE + 4);

  char tmp[512];
  sprintf(tmp,"GPU #%d %s (%dx%d cores) Grid(%dx%d)",
    gpuId,deviceProp.name,deviceProp.multiProcessorCount,
    _ConvertSMVer2Cores(deviceProp.major,deviceProp.minor),
    nbThread / nbThreadPerGroup,
    nbThreadPerGroup);
  deviceName = std::string(tmp);

  // Prefer L1 (We do not use __shared__ at all)
  err = hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  if(err != hipSuccess) {
    printf("GPUEngine: %s\n",hipGetErrorString(err));
    return;
  }

  // Allocate memory
  inputKangaroo = NULL;
  inputKangarooPinned = NULL;
  outputItem = NULL;
  outputItemPinned = NULL;
  jumpPinned = NULL;

  // Input kangaroos
  kangarooSize = nbThread * GPU_GRP_SIZE * KSIZE * 8;
  err = hipMalloc((void **)&inputKangaroo,kangarooSize);
  if(err != hipSuccess) {
    printf("GPUEngine: Allocate input memory: %s\n",hipGetErrorString(err));
    return;
  }
  kangarooSizePinned = nbThreadPerGroup * GPU_GRP_SIZE *  KSIZE * 8;
  err = hipHostAlloc(&inputKangarooPinned,kangarooSizePinned,hipHostMallocWriteCombined | hipHostMallocMapped);
  if(err != hipSuccess) {
    printf("GPUEngine: Allocate input pinned memory: %s\n",hipGetErrorString(err));
    return;
  }

  // OutputHash
  err = hipMalloc((void **)&outputItem,outputSize);
  if(err != hipSuccess) {
    printf("GPUEngine: Allocate output memory: %s\n",hipGetErrorString(err));
    return;
  }
  err = hipHostAlloc(&outputItemPinned,outputSize,hipHostMallocMapped);
  if(err != hipSuccess) {
    printf("GPUEngine: Allocate output pinned memory: %s\n",hipGetErrorString(err));
    return;
  }

  // Jump array
  jumpSize = NB_JUMP * 8 * 4;
  err = hipHostAlloc(&jumpPinned,jumpSize,hipHostMallocMapped);
  if(err != hipSuccess) {
    printf("GPUEngine: Allocate jump pinned memory: %s\n",hipGetErrorString(err));
    return;
  }

  lostWarning = false;
  initialised = true;
  wildOffset.SetInt32(0);

#ifdef GPU_CHECK

  double minT = 1e9;
  for(int i=0;i<5;i++) {
    double t0 = Timer::get_tick();
    check_gpu<<<1,1>>>();
    hipDeviceSynchronize();
    double t1 = Timer::get_tick();
    if( (t1-t0)<minT ) minT = (t1-t0);
  }
  printf("Cuda: %.3f ms\n",minT*1000.0);
  exit(0);

#endif

}

GPUEngine::~GPUEngine() {

  if(inputKangaroo) hipFree(inputKangaroo);
  if(outputItem) hipFree(outputItem);
  if(inputKangarooPinned) hipHostFree(inputKangarooPinned);
  if(outputItemPinned) hipHostFree(outputItemPinned);
  if(jumpPinned) hipHostFree(jumpPinned);

}


int GPUEngine::GetMemory() {
  return kangarooSize + outputSize + jumpSize;
}


int GPUEngine::GetGroupSize() {
  return GPU_GRP_SIZE;
}

bool GPUEngine::GetGridSize(int gpuId,int *x,int *y) {

  if(*x <= 0 || *y <= 0) {

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if(error_id != hipSuccess) {
      printf("GPUEngine: CudaGetDeviceCount %s\n",hipGetErrorString(error_id));
      return false;
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if(deviceCount == 0) {
      printf("GPUEngine: There are no available device(s) that support CUDA\n");
      return false;
    }

    if(gpuId >= deviceCount) {
      printf("GPUEngine::GetGridSize() Invalid gpuId\n");
      return false;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,gpuId);

    if(*x <= 0) *x = 2 * deviceProp.multiProcessorCount;
    if(*y <= 0) *y = 2 * _ConvertSMVer2Cores(deviceProp.major,deviceProp.minor);
    if(*y <= 0) *y = 128;

  }

  return true;

}

void *GPUEngine::AllocatePinnedMemory(size_t size) {

  void *buff;

  hipError_t err = hipHostAlloc(&buff,size,hipHostMallocPortable);
  if(err != hipSuccess) {
    printf("GPUEngine: AllocatePinnedMemory: %s\n",hipGetErrorString(err));
    return NULL;
  }

  return buff;

}

void GPUEngine::FreePinnedMemory(void *buff) {
  hipHostFree(buff);
}

void GPUEngine::PrintCudaInfo() {

  hipError_t err;

  const char *sComputeMode[] =
  {
    "Multiple host threads",
    "Only one host thread",
    "No host thread",
    "Multiple process threads",
    "Unknown",
    NULL
  };

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if(error_id != hipSuccess) {
    printf("GPUEngine: CudaGetDeviceCount %s\n",hipGetErrorString(error_id));
    return;
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if(deviceCount == 0) {
    printf("GPUEngine: There are no available device(s) that support CUDA\n");
    return;
  }

  for(int i = 0; i<deviceCount; i++) {

    err = hipSetDevice(i);
    if(err != hipSuccess) {
      printf("GPUEngine: hipSetDevice(%d) %s\n",i,hipGetErrorString(err));
      return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,i);
    printf("GPU #%d %s (%dx%d cores) (Cap %d.%d) (%.1f MB) (%s)\n",
      i,deviceProp.name,deviceProp.multiProcessorCount,
      _ConvertSMVer2Cores(deviceProp.major,deviceProp.minor),
      deviceProp.major,deviceProp.minor,(double)deviceProp.totalGlobalMem / 1048576.0,
      sComputeMode[deviceProp.computeMode]);

  }

}

int GPUEngine::GetNbThread() {
  return nbThread;
}

void GPUEngine::SetKangaroos(Int *px,Int *py,Int *d) {

  // Sets the kangaroos of each thread
  int gSize = KSIZE * GPU_GRP_SIZE;
  int strideSize = nbThreadPerGroup * KSIZE;
  int nbBlock = nbThread / nbThreadPerGroup;
  int blockSize = nbThreadPerGroup * gSize;
  int idx = 0;

  for(int b = 0; b < nbBlock; b++) {
    for(int g = 0; g < GPU_GRP_SIZE; g++) {
      for(int t = 0; t < nbThreadPerGroup; t++) {

        // X
        inputKangarooPinned[g * strideSize + t + 0 * nbThreadPerGroup] = px[idx].bits64[0];
        inputKangarooPinned[g * strideSize + t + 1 * nbThreadPerGroup] = px[idx].bits64[1];
        inputKangarooPinned[g * strideSize + t + 2 * nbThreadPerGroup] = px[idx].bits64[2];
        inputKangarooPinned[g * strideSize + t + 3 * nbThreadPerGroup] = px[idx].bits64[3];

        // Y
        inputKangarooPinned[g * strideSize + t + 4 * nbThreadPerGroup] = py[idx].bits64[0];
        inputKangarooPinned[g * strideSize + t + 5 * nbThreadPerGroup] = py[idx].bits64[1];
        inputKangarooPinned[g * strideSize + t + 6 * nbThreadPerGroup] = py[idx].bits64[2];
        inputKangarooPinned[g * strideSize + t + 7 * nbThreadPerGroup] = py[idx].bits64[3];

        // Distance
        Int dOff;
        dOff.Set(&d[idx]);
        if(idx % 2 == WILD) dOff.ModAddK1order(&wildOffset);
        inputKangarooPinned[g * strideSize + t + 8 * nbThreadPerGroup] = dOff.bits64[0];
        inputKangarooPinned[g * strideSize + t + 9 * nbThreadPerGroup] = dOff.bits64[1];

#ifdef USE_SYMMETRY
        // Last jump
        inputKangarooPinned[t + 10 * nbThreadPerGroup] = (uint64_t)NB_JUMP;
#endif

        idx++;
      }

    }

    uint32_t offset = b * blockSize;
    hipMemcpy(inputKangaroo + offset,inputKangarooPinned,kangarooSizePinned,hipMemcpyHostToDevice);

  }

  hipError_t err = hipGetLastError();
  if(err != hipSuccess) {
    printf("GPUEngine: SetKangaroos: %s\n",hipGetErrorString(err));
  }

}

void GPUEngine::GetKangaroos(Int *px,Int *py,Int *d) {

  if(inputKangarooPinned==NULL ) {
    printf("GPUEngine: GetKangaroos: Cannot retreive kangaroos, mem has been freed\n");
    return;
  }

  // Sets the kangaroos of each thread
  int gSize = KSIZE * GPU_GRP_SIZE;
  int strideSize = nbThreadPerGroup * KSIZE;
  int nbBlock = nbThread / nbThreadPerGroup;
  int blockSize = nbThreadPerGroup * gSize;
  int idx = 0;

  for(int b = 0; b < nbBlock; b++) {

    uint32_t offset = b * blockSize;
    hipMemcpy(inputKangarooPinned,inputKangaroo + offset,kangarooSizePinned,hipMemcpyDeviceToHost);

    for(int g = 0; g < GPU_GRP_SIZE; g++) {

      for(int t = 0; t < nbThreadPerGroup; t++) {

        // X
        px[idx].bits64[0] = inputKangarooPinned[g * strideSize + t + 0 * nbThreadPerGroup];
        px[idx].bits64[1] = inputKangarooPinned[g * strideSize + t + 1 * nbThreadPerGroup];
        px[idx].bits64[2] = inputKangarooPinned[g * strideSize + t + 2 * nbThreadPerGroup];
        px[idx].bits64[3] = inputKangarooPinned[g * strideSize + t + 3 * nbThreadPerGroup];
        px[idx].bits64[4] = 0;

        // Y
        py[idx].bits64[0] = inputKangarooPinned[g * strideSize + t + 4 * nbThreadPerGroup];
        py[idx].bits64[1] = inputKangarooPinned[g * strideSize + t + 5 * nbThreadPerGroup];
        py[idx].bits64[2] = inputKangarooPinned[g * strideSize + t + 6 * nbThreadPerGroup];
        py[idx].bits64[3] = inputKangarooPinned[g * strideSize + t + 7 * nbThreadPerGroup];
        py[idx].bits64[4] = 0;

        // Distance
        Int dOff;
        dOff.SetInt32(0);
        dOff.bits64[0] = inputKangarooPinned[g * strideSize + t + 8 * nbThreadPerGroup];
        dOff.bits64[1] = inputKangarooPinned[g * strideSize + t + 9 * nbThreadPerGroup];
        if(idx % 2 == WILD) dOff.ModSubK1order(&wildOffset);
        d[idx].Set(&dOff);

        idx++;
      }

    }
  }

  hipError_t err = hipGetLastError();
  if(err != hipSuccess) {
    printf("GPUEngine: GetKangaroos: %s\n",hipGetErrorString(err));
  }

}

void GPUEngine::SetKangaroo(uint64_t kIdx,Int *px,Int *py,Int *d) {

  int gSize = KSIZE * GPU_GRP_SIZE;
  int strideSize = nbThreadPerGroup * KSIZE;
  int blockSize = nbThreadPerGroup * gSize;

  uint64_t t = kIdx % nbThreadPerGroup;
  uint64_t g = (kIdx / nbThreadPerGroup) % GPU_GRP_SIZE;
  uint64_t b = kIdx / (nbThreadPerGroup*GPU_GRP_SIZE);

  // X
  inputKangarooPinned[0] = px->bits64[0];
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 0 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);
  inputKangarooPinned[0] = px->bits64[1];
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 1 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);
  inputKangarooPinned[0] = px->bits64[2];
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 2 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);
  inputKangarooPinned[0] = px->bits64[3];
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 3 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);

  // Y
  inputKangarooPinned[0] = py->bits64[0];
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 4 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);
  inputKangarooPinned[0] = py->bits64[1];
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 5 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);
  inputKangarooPinned[0] = py->bits64[2];
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 6 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);
  inputKangarooPinned[0] = py->bits64[3];
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 7 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);

  // D
  Int dOff;
  dOff.Set(d);
  if(kIdx % 2 == WILD) dOff.ModAddK1order(&wildOffset);
  inputKangarooPinned[0] = dOff.bits64[0];
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 8 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);
  inputKangarooPinned[0] = dOff.bits64[1];
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 9 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);

#ifdef USE_SYMMETRY
  // Last jump
  inputKangarooPinned[0] = (uint64_t)NB_JUMP;
  hipMemcpy(inputKangaroo + (b * blockSize + g * strideSize + t + 10 * nbThreadPerGroup),inputKangarooPinned,8,hipMemcpyHostToDevice);
#endif

}

bool GPUEngine::callKernel() {

  // Reset nbFound
  hipMemset(outputItem,0,4);

  // Call the kernel (Perform STEP_SIZE keys per thread)
  comp_kangaroos << < nbThread / nbThreadPerGroup,nbThreadPerGroup >> >
      (inputKangaroo,maxFound,outputItem,dpMask);

  hipError_t err = hipGetLastError();
  if(err != hipSuccess) {
    printf("GPUEngine: Kernel: %s\n",hipGetErrorString(err));
    return false;
  }

  return true;

}

void GPUEngine::SetParams(uint64_t dpMask,Int *distance,Int *px,Int *py) {
  
  this->dpMask = dpMask;

  for(int i=0;i< NB_JUMP;i++)
    memcpy(jumpPinned + 2*i,distance[i].bits64,16);
  hipMemcpyToSymbol(HIP_SYMBOL(jD),jumpPinned,jumpSize/2);
  hipError_t err = hipGetLastError();
  if(err != hipSuccess) {
    printf("GPUEngine: SetParams: Failed to copy to constant memory: %s\n",hipGetErrorString(err));
    return;
  }

  for(int i = 0; i < NB_JUMP; i++)
    memcpy(jumpPinned + 4 * i,px[i].bits64,32);
  hipMemcpyToSymbol(HIP_SYMBOL(jPx),jumpPinned,jumpSize);
  err = hipGetLastError();
  if(err != hipSuccess) {
    printf("GPUEngine: SetParams: Failed to copy to constant memory: %s\n",hipGetErrorString(err));
    return;
  }

  for(int i = 0; i < NB_JUMP; i++)
    memcpy(jumpPinned + 4 * i,py[i].bits64,32);
  hipMemcpyToSymbol(HIP_SYMBOL(jPy),jumpPinned,jumpSize);
  err = hipGetLastError();
  if(err != hipSuccess) {
    printf("GPUEngine: SetParams: Failed to copy to constant memory: %s\n",hipGetErrorString(err));
    return;
  }

}

bool GPUEngine::callKernelAndWait() {

  // Debug function
  callKernel();
  hipMemcpy(outputItemPinned,outputItem,outputSize,hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  if(err != hipSuccess) {
    printf("GPUEngine: callKernelAndWait: %s\n",hipGetErrorString(err));
    return false;
  }

  return true;

}

bool GPUEngine::Launch(std::vector<ITEM> &hashFound,bool spinWait) {


  hashFound.clear();

  // Get the result

  if(spinWait) {

    hipMemcpy(outputItemPinned,outputItem,outputSize,hipMemcpyDeviceToHost);

  } else {

    // Use hipMemcpyAsync to avoid default spin wait of hipMemcpy wich takes 100% CPU
    hipEvent_t evt;
    hipEventCreate(&evt);
    hipMemcpyAsync(outputItemPinned,outputItem,4,hipMemcpyDeviceToHost,0);
    hipEventRecord(evt,0);
    while(hipEventQuery(evt) == hipErrorNotReady) {
      // Sleep 1 ms to free the CPU
      Timer::SleepMillis(1);
    }
    hipEventDestroy(evt);

  }

  hipError_t err = hipGetLastError();
  if(err != hipSuccess) {
    printf("GPUEngine: Launch: %s\n",hipGetErrorString(err));
    return false;
  }

  // Look for prefix found
  uint32_t nbFound = outputItemPinned[0];
  if(nbFound > maxFound) {
    // prefix has been lost
    if(!lostWarning) {
      printf("\nWarning, %d items lost\nHint: Search with less threads (-g) or increse dp (-d)\n",(nbFound - maxFound));
      lostWarning = true;
    }
    nbFound = maxFound;
  }

  // When can perform a standard copy, the kernel is eneded
  hipMemcpy(outputItemPinned,outputItem,nbFound*ITEM_SIZE + 4,hipMemcpyDeviceToHost);

  for(uint32_t i = 0; i < nbFound; i++) {
    uint32_t *itemPtr = outputItemPinned + (i*ITEM_SIZE32 + 1);
    ITEM it;

    it.kIdx = *((uint64_t*)(itemPtr + 12));

    uint64_t *x = (uint64_t *)itemPtr;
    it.x.bits64[0] = x[0];
    it.x.bits64[1] = x[1];
    it.x.bits64[2] = x[2];
    it.x.bits64[3] = x[3];
    it.x.bits64[4] = 0;

    uint64_t *d = (uint64_t *)(itemPtr + 8);
    it.d.bits64[0] = d[0];
    it.d.bits64[1] = d[1];
    it.d.bits64[2] = 0;
    it.d.bits64[3] = 0;
    it.d.bits64[4] = 0;
    if(it.kIdx % 2 == WILD) it.d.ModSubK1order(&wildOffset);

    hashFound.push_back(it);
  }

  return callKernel();

}
